/* 
* Solves the Panfilov model using an explicit numerical scheme.
* Based on code orginally provided by Xing Cai, Simula Research Laboratory
*and reimplementation by Scott B. Baden, UCSD
*
* Modified and restructured by Didem Unat, Koc Universiy_thread
*
* Refer to "Detailed Numerical Analyses of the Aliev-Panfilov Model on GPGPU"
* https://www.simula.no/publications/detailed-numerical-analyses-aliev-panfilov-model-gpgpu
*by Xing Cai, Didem Unat and Scott Baden
*
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <iostream>
#include <iomanip>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <getopt.h>

using namespace std;

// External functions
extern "C" void splot(double *E, double T, int niter, int m, int n);

void cmdLine(int argc, char* argv[], double& T, int& n, int& px, int& py, int& plot_freq, int& kernel_no);

// Utilities
// 

// Timer
// Make successive calls and take a difference to get the elapsed time.
static const double kMicro = 1.0e-6;
double getTime()
{
    struct timeval TV;
    struct timezone TZ;

    const int RC = gettimeofday(&TV, &TZ);
    if (RC == -1) {
        cerr << "ERROR: Bad call to gettimeofday" << endl;
        return(-1);
    }

    return(((double)TV.tv_sec) + kMicro * ((double)TV.tv_usec));

}  // end getTime()

// Allocate a 2D array
double **alloc2D(int m, int n) {
    double **E;
    int nx = n, ny = m;
    E = (double**)malloc(sizeof(double*) * ny + sizeof(double) * nx * ny);
    assert(E);
    int j;
    for (j = 0;j < ny;j++)
        E[j] = (double*)(E + ny) + j * nx;
    return(E);
}

// Reports statistics about the computation
// These values should not vary (except to within roundoff)
// when we use different numbers of  processes to solve the problem
double stats(double *E, int m, int n, double *_mx) {
	double mx = -1;
	double l2norm = 0;
	int i, j;
	for (j = 1; j <= m; j++) {
	  for (i = 1; i <= n; i++) {
		  l2norm += E[j * (n+2) + i] * E[j * (n+2) + i];
		  if (E[j * (n+2) + i] > mx) 
              mx = E[j * (n+2) + i];
			}
	}
	*_mx = mx;
	l2norm /= (double) ((m) * (n));
	l2norm = sqrt(l2norm);
	return l2norm;
}



__global__ void ghostKernel(double *E_prev, const int n, const int m) {
    int j = threadIdx.x + 1;

    E_prev[j * (n+2)] = E_prev[j * (n+2) + 2];
    E_prev[j * (n+2) + (n + 1)] = E_prev[j * (n + 2) + (n - 1)];

    E_prev[j] = E_prev[2 * (n + 2) + j];
    E_prev[(m + 1) * (n + 2) + j] = E_prev[(m - 1) * (n + 2) + j];
}

__global__ void singleKernel(double *E, double *E_prev, double *R,
    const int n, const int m, const double kk,
    const double dt, const double a, const double epsilon,
    const double M1, const double M2, const double b, const double alpha) {
   
    int x_thread = threadIdx.x, y_thread = threadIdx.y, x_block = blockIdx.x, y_block = blockIdx.y, x_blockDim = blockDim.x, y_blockDim = blockDim.y;
   const int block_size = 16;
    
    __shared__ double device_memory_array[block_size + 2][block_size + 2];     
    
    if(x_thread == 0) {
        int index = (y_block * y_blockDim * (n + 2)) + (x_block * x_blockDim) + ((y_thread + 1) * (n + 2));
        for (int j = 0; j < x_blockDim + 2; j++) {
            device_memory_array[y_thread + 1][j] = E_prev[index + j];
        }
        if(y_thread == 0) {
            int index = (y_block * y_blockDim * (n + 2)) + (x_block * x_blockDim);    
            for (int j = 0; j < x_blockDim + 2; j++) {
                device_memory_array[0][j] = E_prev[index + j];
            }
        }
        if(y_thread == 1) {
            int index = (y_block * y_blockDim * (n + 2)) + (x_block * x_blockDim) + ((y_blockDim + 1) * (n + 2));
    
            for (int j = 0; j < x_blockDim + 2; j++) {
                device_memory_array[y_blockDim + 1][j] = E_prev[index + j];
            }
        }
    }
    
    int index = (y_block * y_blockDim * (n + 2)) + (x_block * x_blockDim) + (n + 2) + 1 + (y_thread * (n + 2) + x_thread);
    
    __syncthreads();
    double E_temp = E[index]; 
    double R_temp = R[index];
     
    E_temp = device_memory_array[y_thread + 1][x_thread + 1] + alpha * (device_memory_array[y_thread + 1][x_thread + 2] + device_memory_array[y_thread + 1][x_thread] - 4 * device_memory_array[y_thread + 1][x_thread + 1] + device_memory_array[y_thread + 2][x_thread + 1] + device_memory_array[y_thread][x_thread + 1]);
    E_temp = E_temp - dt * (kk * E_temp * (E_temp - a) * (E_temp - 1) + E_temp * R_temp);
    R_temp = R_temp + dt * (epsilon + M1 * R_temp / (E_temp + M2)) * (-R_temp - kk * E_temp * (E_temp - b - 1));

    __syncthreads();
    E[index] = E_temp;
    R[index] = R_temp;
 }

void simulate(double *E, double *E_prev, double *R,
              const double alpha, const int n, const int m, const double kk,
              const double dt, const double a, const double epsilon,
              const double M1, const double M2, const double b) {
    
    const dim3 block_size(16,16);
    const dim3 num_blocks(n / block_size.x, n / block_size.y);

    ghostKernel<<<1, n>>>(E_prev, n, m);
    singleKernel<<<num_blocks, block_size>>>(E, E_prev, R, n, m, kk, dt, a, epsilon, M1, M2, b, alpha);
}

// Main program
int main(int argc, char** argv)
{
    /*
     *  Solution arrays
     *   E is the "Excitation" variable, a voltage
     *   R is the "Recovery" variable
     *   E_prev is the Excitation variable for the previous timestep,
     *      and is used in time integration
     */
    double *E, *R, *E_prev;
    double *d_E, *d_R, *d_E_prev;

    // Various constants - these definitions shouldn't change
    const double a = 0.1, b = 0.1, kk = 8.0, M1 = 0.07, M2 = 0.3, epsilon = 0.01, d = 5e-5;

    double T = 1000.0;
    int m = 200, n = 200;
    int plot_freq = 0;
    int bx = 1, by = 1;
    int kernel = 1;

    cmdLine(argc, argv, T, n, bx, by, plot_freq, kernel);
    m = n;
    // Allocate contiguous memory for solution arrays
    // The computational box is defined on [1:m+1,1:n+1]
    // We pad the arrays in order to facilitate differencing on the 
    // boundaries of the computation box
    E = (double *) malloc(sizeof(double) * size_t((m + 2) * (n + 2)));
    E_prev = (double *) malloc(sizeof(double) * size_t((m + 2) * (n + 2)));
    R = (double *) malloc(sizeof(double) * size_t((m + 2) * (n + 2)));

    int i, j;
    // Initialization
    for (j = 1; j <= m; j++)
        for (i = 1; i <= n; i++)
            E_prev[j * (n+2) + i] = R[j * (n+2) + i] = 0;

    for (j = 1; j <= m; j++)
        for (i = n / 2 + 1; i <= n; i++)
            E_prev[j * (n+2) + i] = 1.0;

    for (j = m / 2 + 1; j <= m; j++)
        for (i = 1; i <= n; i++)
            R[j * (n+2) + i] = 1.0;

    double dx = 1.0 / n;

    // For time integration, these values shouldn't change 
    double rp = kk * (b + 1) * (b + 1) / 4;
    double dte = (dx * dx) / (d * 4 + ((dx * dx)) * (rp + kk));
    double dtr = 1 / (epsilon + ((M1 / M2) * rp));
    double dt = (dte < dtr) ? 0.95 * dte : 0.95 * dtr;
    double alpha = d * dt / (dx * dx);

    cout << "Grid Size       : " << n << endl;
    cout << "Duration of Sim : " << T << endl;
    cout << "Time step dt    : " << dt << endl;
    cout << "Block Size: " << bx << " x " << by << endl;
    cout << "Using CUDA Kernel Version: " << kernel << endl;

    cout << endl;

    // Start the timer
    double t0 = getTime();


    // Simulated time is different from the integer timestep number
    // Simulated time
    double t = 0.0;
    // Integer timestep number
    int niter = 0;

    hipMalloc((void **) &d_E, sizeof(double) * (m + 2) * (n + 2));
    hipMalloc((void **) &d_E_prev, sizeof(double) * (m + 2) * (n + 2));
	hipMalloc((void **) &d_R, sizeof(double) * (m + 2) * (n + 2));

    hipMemcpy(d_E, E, sizeof(double) * (m + 2) * (n + 2), hipMemcpyHostToDevice);
    hipMemcpy(d_E_prev, E_prev, sizeof(double) * (m + 2) * (n + 2), hipMemcpyHostToDevice);
	hipMemcpy(d_R, R, sizeof(double) * (m + 2) * (n + 2), hipMemcpyHostToDevice);

    while (t < T) {

        t += dt;
        niter++;

        simulate(d_E, d_E_prev, d_R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b);

        //swap current E with previous E
        double *tmp = d_E; d_E = d_E_prev; d_E_prev = tmp;

        if (plot_freq) {
            int k = (int)(t / plot_freq);
            if ((t - k * plot_freq) < dt) {
                hipMemcpy(E, d_E, sizeof(double) * (m + 2) * (n + 2), hipMemcpyDeviceToHost);
                splot(E, t, niter, m + 2, n + 2);
            }
        }
    }//end of while loop

    hipMemcpy(E_prev, d_E_prev, sizeof(double) * (m + 2) * (n + 2), hipMemcpyDeviceToHost);

    hipFree(d_E);
    hipFree(d_E_prev);
    hipFree(d_R);  

    double time_elapsed = getTime() - t0;

    double Gflops = (double)(niter * (1E-9 * n * n) * 28.0) / time_elapsed;
    double BW = (double)(niter * 1E-9 * (n * n * sizeof(double) * 4.0)) / time_elapsed;

    cout << "Number of Iterations        : " << niter << endl;
    cout << "Elapsed Time (sec)          : " << time_elapsed << endl;
    cout << "Sustained Gflops Rate       : " << Gflops << endl;
    cout << "Sustained Bandwidth (GB/sec): " << BW << endl << endl;

    double mx;
    double l2norm = stats(E_prev, m, n, &mx);
    cout << "Max: " << mx << " L2norm: " << l2norm << endl;

    if (plot_freq) {
        cout << "\n\nEnter any input to close the program and the plot..." << endl;
        getchar();
    }

    free(E);
    free(E_prev);
    free(R);

    return 0;
}

void cmdLine(int argc, char* argv[], double& T, int& n, int& bx, int& by, int& plot_freq, int& kernel) {
    /// Command line arguments
     // Default value of the domain sizes
    static struct option long_options[] = {
           {"n", required_argument, 0, 'n'},
           {"bx", required_argument, 0, 'x'},
           {"by", required_argument, 0, 'y'},
           {"tfinal", required_argument, 0, 't'},
           {"plot", required_argument, 0, 'p'},
       {"kernel_version", required_argument, 0, 'v'},
    };
    // Process command line arguments
    int ac;
    for (ac = 1;ac < argc;ac++) {
        int c;
        while ((c = getopt_long(argc, argv, "n:x:y:t:p:v:", long_options, NULL)) != -1) {
            switch (c) {

                // Size of the computational box
            case 'n':
                n = atoi(optarg);
                break;

                // X block geometry
            case 'x':
                bx = atoi(optarg);

                // Y block geometry
            case 'y':
                by = atoi(optarg);

                // Length of simulation, in simulated time units
            case 't':
                T = atof(optarg);
                break;

                // Plot the excitation variable
            case 'p':
                plot_freq = atoi(optarg);
                break;

                // Kernel version
            case 'v':
                kernel = atoi(optarg);
                break;

                // Error
            default:
                printf("Usage:  [-n <domain size>] [-t <final time >]\n\t [-p <plot frequency>]\n\t[-x <x block geometry> [-y <y block geometry][-v <Kernel Version>]\n");
                exit(-1);
            }
        }
    }
}
/* **********************************************************
 *  Author : Urvashi R.V. [04/06/2004]
 *      Modified by Didem Unat [03/23/21]
 *************************************************************/

#include <stdio.h>

 /* Function to plot the 2D array
  * 'gnuplot' is instantiated via a pipe and
  * the values to be plotted are passed through, along
  * with gnuplot commands */

FILE* gnu = NULL;

void splot(double *U, double T, int niter, int m, int n)
{
    int i, j;
    if (gnu == NULL) gnu = popen("gnuplot", "w");

    double mx = -1, mn = 32768;
    for (j = 0; j < m; j++)
        for (i = 0; i < n; i++) {
            if (U[j * m + i] > mx)
                mx = U[j * m + i];
            if (U[j * m + i] < mn)
                mn = U[j * m + i];
        }

    fprintf(gnu, "set title \"T = %f [niter = %d]\"\n", T, niter);
    fprintf(gnu, "set size square\n");
    fprintf(gnu, "set key off\n");
    fprintf(gnu, "set pm3d map\n");
    // Various color schemes
    fprintf(gnu, "set palette defined (-3 \"blue\", 0 \"white\", 1 \"red\")\n");

    //    fprintf(gnu,"set palette rgbformulae 22, 13, 31\n");
    //    fprintf(gnu,"set palette rgbformulae 30, 31, 32\n");

    fprintf(gnu, "splot [0:%d] [0:%d][%f:%f] \"-\"\n", m - 1, n - 1, mn, mx);
    for (j = 0; j < m; j++) {
        for (i = 0; i < n; i++) {
            fprintf(gnu, "%d %d %f\n", i, j, U[i * m + j]);
        }
        fprintf(gnu, "\n");
    }
    fprintf(gnu, "e\n");
    fflush(gnu);
    return;
}
